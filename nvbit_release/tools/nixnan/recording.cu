#include "hip/hip_runtime.h"
#include "recording.h"
#include "utils/utils.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include "common.cuh"

namespace nixnan {
    const char NOTYPE = 127;
    recorder::recorder(size_t sz) : size(sz), current_entry(0) {
        size_t sh_amt = SH_AMT;
        if (sz > (1ULL << (32 - sh_amt))) {
            throw std::runtime_error("Requested log size is too large");
        }
        inst_info = new host_entry[sz];
        CUDA_SAFECALL(hipMalloc((void**)&device_errors, sz * sizeof(uint32_t) << sh_amt));
        host_errors = new uint32_t[sz << sh_amt];
        for(size_t i = 0; i < sz << sh_amt; i++) {
            host_errors[i] = 0;
        }
        CUDA_SAFECALL(hipMemcpy(device_errors, host_errors, sz * sizeof(uint32_t) << sh_amt, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }
    recorder::~recorder() {
        delete[] inst_info;
        if (host_errors) {
            delete[] host_errors;
        }
        if (device_errors) {
            CUDA_SAFECALL(hipFree(device_errors));
            hipDeviceSynchronize();
        }
    }
    uint32_t recorder::mk_entry(Instr *instr, const std::vector<std::pair<reginfo, std::vector<reginsertion>>> &regs, hipCtx_t ctx, hipFunction_t f) {
        std::string instr_str = instr->getSass();
        uint32_t offset = instr->getOffset();
        char *file_name = (char *)malloc(sizeof(char) * 1024);
        file_name[0] = '\0';
        char *dir_name = (char *)malloc(sizeof(char) * 1024);
        dir_name[0] = '\0';
        uint32_t line = 0;
        bool ret_line_info = nvbit_get_line_info(ctx, f, offset, &file_name, &dir_name, &line);
        std::string path = file_name;
        path += dir_name;
        std::string line_str = std::to_string(line);
        std::string func = cut_kernel_name(nvbit_get_func_name(ctx, f));
        free(file_name);
        free(dir_name);
        char optypes[OPERANDS] = {NOTYPE};
        for (size_t i = 0; i < regs.size(); i++) {
            optypes[i] = regs[i].first.type;
        }
        return mk_entry(instr_str, path, line_str, func, optypes);
    }
    uint32_t recorder::mk_entry(std::string& instr, std::string& path, std::string& line, std::string& func, char* optypes) {
        inst_info[current_entry].instr = instr;
        inst_info[current_entry].path = path;
        inst_info[current_entry].line = line;
        inst_info[current_entry].func = func;
        std::memcpy(inst_info[current_entry].opertypes, optypes, OPERANDS);
        return current_entry++;
    }
    void recorder::free_device() {
        CUDA_SAFECALL(hipFree(device_errors));
        device_errors = nullptr;
    }
    void recorder::end() {
        CUDA_SAFECALL(hipMemcpy(host_errors, device_errors, size * sizeof(uint32_t) << SH_AMT, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
    }
    device_recorder recorder::get_device_recorder() {
        return device_recorder(device_errors);
    }
    std::string& recorder::get_inst(uint32_t id) {
        return inst_info[id].instr;
    }
    std::string& recorder::get_path(uint32_t id) {
        return inst_info[id].path;
    }
    std::string& recorder::get_line(uint32_t id) {
        return inst_info[id].line;
    }
    std::string& recorder::get_func(uint32_t id) {
        return inst_info[id].func;
    }
    uint32_t recorder::get_type(uint32_t id, uint32_t op) {
        return inst_info[id].opertypes[op];
    }
    uint32_t recorder::get_exce(uint32_t id, uint32_t exce, uint32_t op) {
        size_t index = id << (SH_AMT) | op << EXCEBITS | exce;
        return host_errors[index];
    }
}