#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* every tool needs to include this once */
#include "nvbit_tool.h"
/* nvbit interface file */
#include "nvbit.h"
/* for channel */
#include "utils/channel.hpp"

#include <unordered_set>
#include <memory>
#include <thread>

#include "recording.h"
#include "exception_info.cuh"
using nixnan::exception_info;
#include "common.cuh"
#include "instruction_info.cuh"

uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int func_details = 0;
int print_ill_instr = 0;
int sampling = 0;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

#define CHANNEL_SIZE (1l << 10)
#define TABLE_SIZE (1l << 17)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

std::unordered_set<std::string> kernel_whitelist;
std::unordered_set<std::string> kernel_blacklist;
std::unordered_map<std::string, int> analyzed_kernels;

// pthread_t recv_thread;
std::thread recv_thread;
std::shared_ptr<nixnan::recorder> recorder = nullptr;
std::unordered_set<hipFunction_t> instrumented_functions;

bool skip_flag = false;

void nvbit_at_init() {
  // Disable warning about using CUDA API calls in nvbit_at_init.
  setenv("ACK_CTX_INIT_LIMITATION", "1", 1);
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
              "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  GET_VAR_INT(func_details, "ENABLE_FUN_DETAIL", 0,
              "Enable detailed function information for kernel");
  GET_VAR_INT(print_ill_instr, "PRINT_ILL_INSTR", 0,
              "Print the instruction which caused the exception");
  GET_VAR_INT(
      sampling, "SAMPLING", 0,
      "Instrument a repeat kernel every SAMPLING times");
  std::string pad(82, '-');
  std::cerr << pad << '\n';
}


void instrument_function(hipCtx_t ctx, hipFunction_t func) {
  /* Get related functions of the kernel (device function that can be
   * called by the kernel) */
  std::vector<hipFunction_t> related_functions =
      nvbit_get_related_functions(ctx, func);
  related_functions.push_back(func);

  for (auto f : related_functions) {
    if (!instrumented_functions.insert(f).second) {
      continue;
    }

    std::string kname = cut_kernel_name(nvbit_get_func_name(ctx, func));
    if (verbose) {
      std::cerr << "#nixnan: Inspecting function " << nvbit_get_func_name(ctx, f) <<
                   " at address 0x" << std::hex << nvbit_get_func_addr(f) << std::endl;
    }

    for (auto instr : nvbit_get_instrs(ctx, func)){
      auto reginfo = instruction_info::get_reginfo(instr);
      if (reginfo.empty()) { continue; }
      if (verbose) {
        std::cerr << "#nixnan: Instrumenting instruction " << instr->getSass() << std::endl;
      }
      uint32_t reg_num = 0;
      nvbit_insert_call(instr, "nixnan_check_regs", IPOINT_AFTER);
      nvbit_add_call_arg_guard_pred_val(instr);
      nvbit_add_call_arg_const_val64(instr, tobits64(recorder->get_device_recorder()), false);
      uint32_t inst_id = recorder->mk_entry(instr, reginfo[0].first.type, ctx, f);
      // std::cerr << "#nixnan: Instrumenting instruction with ID " << inst_id << std::endl;
      nvbit_add_call_arg_const_val32(instr, inst_id, false);
      nvbit_add_call_arg_const_val64(instr, tobits64(&channel_dev), false);
      for (auto [ri, rfuns] : reginfo) {
        if (reg_num > 0 && !ri.div0) continue;
        nvbit_add_call_arg_const_val32(instr, 1 + rfuns.size());
        nvbit_add_call_arg_const_val32(instr, tobits32(ri), true);
        for (auto& rfun : rfuns) {
          rfun();
        }
        reg_num++;
      }
    }
  }
}

// Kernel to run to flush the rest of the channel
__global__ void flush_channel() {
  // Push negative cta information to the channel to indicate the end of execution
  exception_info ei(int4{-1, -1, -1, -1}, 0, 0, 0);
  // Generates the following warning:
  // /.../device_atomic_functions.hpp(196): Warning: Cannot do atomic on local memory
  // This is inside the nvbit library.
  channel_dev.push(&ei, sizeof(exception_info));
  /* flush channel */
  channel_dev.flush();
}

void recv_thread_fun(std::shared_ptr<nixnan::recorder> recorder, ChannelHost channel_host) {
  char *recv_buffer = new char[CHANNEL_SIZE];

  while (recv_thread_started) {
    uint32_t num_recv_bytes = 0;

    if (recv_thread_receiving &&
        (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
      uint32_t num_processed_bytes = 0;
      while (num_processed_bytes < num_recv_bytes) {
        exception_info *ei = reinterpret_cast<exception_info*>(&recv_buffer[num_processed_bytes]);
        /* when we get this cta_id_x it means the kernel has completed
          */
        if (ei->warp() == -1) {
          recv_thread_receiving = false;
          break;
        }
        uint32_t id = ei->inst();
        std::string instr = recorder->get_inst(id);
        std::string func = recorder->get_func(id);
        std::string path = recorder->get_path(id);
        std::string line = recorder->get_line(id);
        std::string type = type_to_string.at(recorder->get_type(id));

        uint32_t exce = ei->exception();
        std::vector<std::string> exceptions;
        if (exce & E_NAN) {
          exceptions.push_back("NaN");
        }
        if (exce & E_INF) {
          exceptions.push_back("infinity");
        }
        if (exce & E_SUB) {
          exceptions.push_back("subnormal");
        }
        if (exce & E_DIV0) {
          exceptions.push_back("div0");
        }
        std::string errors;
        for (size_t i = 0; i < exceptions.size(); ++i) {
          errors += exceptions[i];
          if (i != exceptions.size() - 1) errors += ",";
        }
        std::cerr << "#nixnan: error [" << errors << "] detected in instruction " << instr << " in function "
                  << func << " at line " << line << " of type " << type << std::endl;
        num_processed_bytes += sizeof(exception_info);
      }
    }
  }
  delete[] recv_buffer;
  return;
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
  const char *name, void *params, hipError_t *pStatus) {
  if (skip_flag)
    return;

  if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel ||
      cbid == API_CUDA_cuLaunchCooperativeKernel ||
      cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz ||
      cbid == API_CUDA_cuLaunchCooperativeKernelMultiDevice) {
    cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

    if (!is_exit) {
      /*----- Instrumentation Logic --------- */
      std::string kernel_name = nvbit_get_func_name(ctx, p->f);
      std::string short_name = cut_kernel_name(kernel_name);
      bool enable_instr = false;
      recv_thread_receiving = true;

      if (!kernel_whitelist.empty()) {
        enable_instr = kernel_whitelist.count(short_name);
      } else if (!kernel_blacklist.empty()) {
        enable_instr = !kernel_blacklist.count(short_name);
      } else {
        enable_instr = true;
      }

      if (sampling != 0 && analyzed_kernels.count(short_name)) {
        if (analyzed_kernels[short_name] % sampling != 0) {
          ++analyzed_kernels[short_name];
          enable_instr = false;
        }
      }

      if (enable_instr) {
        instrument_function(ctx, p->f);
        // Initialize kernel count if not present, then increment
        int count = analyzed_kernels[short_name]++;
        if (count == 0) {
          std::cerr << "#nixnan: running kernel [" << short_name << "] ..." << std::endl;
        } else if (func_details) {
          std::cout << "#nixnan: running kernel [" << kernel_name << "] ..."
                    << std::endl;
        }
        ++analyzed_kernels[short_name];
      }
      nvbit_enable_instrumented(ctx, p->f, enable_instr);
      /*------------ End of Instrumentation Logic ---------------*/
    } else {
      /* make sure current kernel is completed */
      hipDeviceSynchronize();
      hipError_t kernelError = hipGetLastError();
      if (kernelError != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(kernelError));
        assert(0);
      }

      /* make sure we prevent re-entry on the nvbit_callback when issuing
      * the flush_channel kernel */
      skip_flag = true;

      /* issue flush of channel so we are sure all the memory accesses
      * have been pushed */
      flush_channel<<<1, 1>>>();
      hipDeviceSynchronize();
      assert(hipGetLastError() == hipSuccess);

      /* unset the skip flag */
      skip_flag = false;

      /* wait here until the receiving thread has not finished with the
      * current kernel */
      while (recv_thread_receiving) {
        sched_yield();
      }
    }
  }
}

void nvbit_tool_init(hipCtx_t ctx) {
  std::string k_whitelist_name = "kernel_whitelist.txt";
  std::string k_blacklist_name = "kernel_blacklist.txt";

  std::cerr << "#nixnan: Initializing GPU context...\n";
  kernel_whitelist = read_from_file(k_whitelist_name);
  kernel_blacklist = read_from_file(k_blacklist_name);
  if (!kernel_whitelist.empty()) {
    std::cerr << "#nixnan: only instrumenting kernels specified in "
              << k_whitelist_name << std::endl;
  } else if (!kernel_blacklist.empty()) {
    std::cerr << "#nixnan: not instrumenting kernels specified in "
              << k_blacklist_name << std::endl;
  } else {
    std::cerr << "#nixnan: instrumenting all kernels" << std::endl;
  }
  recorder = std::make_shared<nixnan::recorder>(TABLE_SIZE);
  recv_thread_started = true;
  channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
  recv_thread = std::thread(recv_thread_fun, recorder, channel_host);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  if (recv_thread_started) {
    recv_thread_started = false;
    recv_thread.join();
  }
  recorder->end();
  recorder->free_device();
  size_t num_inst = recorder->get_size();

  uint32_t *host_errors = recorder->get_host_errors();

  std::map<uint32_t,std::array<uint32_t, 4>> exception_counts;
  exception_counts[FP16] = {};
  exception_counts[FP32] = {};
  exception_counts[FP64] = {};

  for (size_t i = 0; i < num_inst; ++i) {
    for (int exce = 0; exce < 16; exce++) {
      size_t idx = i << 4 | exce;
      if (host_errors[idx] == 0) continue;
      uint32_t type = recorder->get_type(i);
      if (exce & E_NAN) {
        exception_counts[type][0] += 1;
      }
      if (exce & E_INF) {
        exception_counts[type][1] += 1;
      }
      if (exce & E_SUB) {
        exception_counts[type][2] += 1;
      }
      if (exce & E_DIV0) {
        exception_counts[type][3] += 1;
      }

    }
  }

  std::cerr << "#nixnan: Finalizing GPU context...\n\n";

  std::cerr << "#nixnan: ------------ nixnan Report -----------\n\n";

  auto print_type_exceptions = [&](const std::string& type_name, uint32_t type_id) {
    std::cerr << "#nixnan: --- " << type_name << " Operations ---\n";
    std::cerr << "#nixnan: NaN: " << exception_counts[type_id][0] << "\n";
    std::cerr << "#nixnan: Infinity: " << exception_counts[type_id][1] << "\n";
    std::cerr << "#nixnan: Subnormal: " << exception_counts[type_id][2] << "\n";
    std::cerr << "#nixnan: Division by 0: " << exception_counts[type_id][3] << "\n\n";
  };

  print_type_exceptions("FP16", FP16);
  print_type_exceptions("FP32", FP32);
  print_type_exceptions("FP64", FP64);
}