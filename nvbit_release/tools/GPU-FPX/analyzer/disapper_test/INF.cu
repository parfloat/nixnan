/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
  if (comp < +1.4882E-36f / (var_2 / var_3)) {
  if (comp < (-1.3710E-36f * (var_4 * asinf(var_5 - (+1.8810E35f * (-1.7451E-36f - -1.8252E25f)))))) {
    for (int i=0; i < var_1; ++i) {
      comp += (-1.2725E20f - var_6);
if (comp == sinhf((var_7 * (var_8 / (var_9 * +1.3792E36f))))) {
  comp += -1.1554E35f / (var_10 - (var_11 / var_12));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        return 1;
    }
  hipDeviceSynchronize();

  return 0;
}
