/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float* var_7,float var_8,float var_9,float var_10) {
if (comp <= asinf(+1.3463E34f)) {
  comp += (var_3 + sinhf(-0.0f));
for (int i=0; i < var_1; ++i) {
  comp += ldexpf(atan2f(+1.5696E-42f, var_4 * var_5 - asinf(+1.2147E35f)), 2);
comp += acosf(logf((+1.6666E-36f / -1.1833E-36f + var_6 + +1.8570E35f)));
}
for (int i=0; i < var_2; ++i) {
  var_7[i] = (var_8 * ceilf(var_9 + logf(var_10 - -1.7568E-37f + -1.7043E26f)));
comp += var_7[i] - floorf(fabsf(+1.2484E35f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
