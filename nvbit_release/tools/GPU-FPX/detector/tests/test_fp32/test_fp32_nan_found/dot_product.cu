

#include <hip/hip_runtime.h>
#include <stdio.h>

// __device__ void mul(float a, float b, float *res)
// {
//   *res = a * b;
//   // NaN
//   *res = (*res)-(*res) / (*res)-(*res);
// }

__global__ void dot_prod(float *x, float *y, int size) {
  float d;
  for (int i = 0; i < size; ++i) {
    float tmp;
    // mul(x[i], y[i], &tmp);
    tmp = x[i] * y[i];
    printf("first tmp is %f\n", tmp);
    // NaN
    tmp = (tmp - tmp) / (tmp - tmp);
    printf("second tmp is %f\n", tmp);
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
