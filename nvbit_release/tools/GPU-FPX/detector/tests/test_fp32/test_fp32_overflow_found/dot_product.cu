

#include <hip/hip_runtime.h>
#include <stdio.h>

// __device__ void mul(float a, float b, float *res)
// {
//   *res = a * b;
//   // Overflow

// #ifdef FPC_POSITIVE_OVERFLOW
//   *res = (*res) * 1e38;
// #else
//   *res = (*res) * 1e38;
// #endif
// }

__global__ void dot_prod(float *x, float *y, int size)
{
  float d;
  for (int i=0; i < size; ++i)
  {
    float tmp;
    // mul(x[i], y[i], &tmp);
    tmp = x[i]*y[i];
  // Overflow

//  #ifdef FPC_POSITIVE_OVERFLOW
 //   tmp = (tmp) * 1e38;
 // #else
    tmp = tmp * 1e38;
 // #endif
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
