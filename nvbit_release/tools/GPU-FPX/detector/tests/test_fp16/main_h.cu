
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h> 
#include <hip/hip_fp16.h>
using namespace std;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {   
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }   
}
__global__ void half_plus1(__half *in_array)
{
    const int idx = threadIdx.x + blockDim.x*blockIdx.x;
    //in_array[idx] = __float2half(__half2float(in_array[idx]) + 1.0);
    in_array[idx] = __hdiv(in_array[idx], 1.5);
}
int main(void)
{
    const int n = 64;
    __half *h_in, *d_in;
    h_in = (__half*) malloc(n*sizeof(__half));
    gpuErrchk( hipMalloc(&d_in, n*sizeof(__half)) );
    for (int i=0; i<n; i++)
        h_in[i] = __float2half(1.5);
    gpuErrchk( hipMemcpy(d_in, h_in, n*sizeof(__half), hipMemcpyHostToDevice) );
    dim3 block_dims(2,1,1);
    dim3 thread_dims(32,1,1);
    half_plus1<<<block_dims, thread_dims>>>(d_in);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpy(h_in, d_in, n*sizeof(__half), hipMemcpyDeviceToHost) );
    for (int i=0; i<n; i++)
    {   
        // if(__half2float(h_in[i]) != 2.5)
        // {
        //     cout<< "Mismatch at " << i << " Expected = 2.5 " << "Actual = " << __half2float(h_in[i]) << endl;
        //     exit(1);
        // }
        cout<<"Actual = " << __half2float(h_in[i]) << endl;
    }    
    //cout << "TEST PASSES" << endl;
    hipFree(d_in);
    free(h_in);
    return 0;
}
