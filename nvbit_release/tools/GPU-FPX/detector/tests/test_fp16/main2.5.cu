
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h> 
#include <hip/hip_fp16.h>
using namespace std;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {   
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }   
}
__global__ void half2_plus1(int n,__half *in_array)
{
    const int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx%2==0&&idx<n) {
        half2 *in_array2 = (half2*)in_array;
        //in_array2[idx/2] = __hadd2(__halves2half2(in_array[idx], in_array[idx+1]),  __halves2half2(__float2half(1.9),__float2half(7.4)));
        in_array2[idx/2] = __hadd2(__halves2half2(in_array[idx], in_array[idx+1]),  __float2half2_rn(1));
    }
}
int main(void)
{
    const int n = 64;
    // __half2 *h_in, *d_in;
    // h_in = (__half2*) malloc(n*sizeof(__half2));
    // gpuErrchk( cudaMalloc(&d_in, n*sizeof(__half2)) );
    // for (int i=0; i<n; i++)
    //     h_in[i] = __float2half2_rn(1.5);
    // gpuErrchk( cudaMemcpy(d_in, h_in, n*sizeof(__half2), cudaMemcpyHostToDevice) );
    // dim3 block_dims(2,1,1);
    // dim3 thread_dims(32,1,1);
    // half2_plus1<<<block_dims, thread_dims>>>(d_in);
    // gpuErrchk( cudaPeekAtLastError() );
    // gpuErrchk( cudaDeviceSynchronize() );
    // gpuErrchk( cudaMemcpy(h_in, d_in, n*sizeof(__half2), cudaMemcpyDeviceToHost) );
    __half *h_in, *d_in;
    h_in = (__half*) malloc(n*sizeof(__half));
    gpuErrchk( hipMalloc(&d_in, n*sizeof(__half)) );
    for (int i=0; i<n; i+=2){
        h_in[i] = __float2half(3.5);
        cout << "h[" << i << "] = " << __half2float(h_in[i]) << endl;
    }
    for (int i=1; i<n; i+=2){
        h_in[i] = __float2half(0.5);
        cout << "h[" << i << "] = " << __half2float(h_in[i]) << endl;
    }
    gpuErrchk( hipMemcpy(d_in, h_in, n*sizeof(__half), hipMemcpyHostToDevice) );
    dim3 block_dims(2,1,1);
    dim3 thread_dims(32,1,1);
    half2_plus1<<<block_dims, thread_dims>>>(n,d_in);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpy(h_in, d_in, n*sizeof(__half), hipMemcpyDeviceToHost) );
    for (int i=0; i<n; i++)
    {   
        // if((__high2float(h_in[i]) != 2.5) || (__low2float(h_in[i]) !=2.5))
        // {
        //     cout<< "Mismatch at " << i << " Expected = 2.5 " << "Actual = " << __half2float(h_in[i].x) << " " << __half2float(h_in[i].y) << endl;
        //     exit(1);
        // }
        cout<< "h_in[" << i << "] = " << __half2float(h_in[i])  << endl;
    }   
    cout << "TEST PASSES" << endl;
    hipFree(d_in);
    free(h_in);
    return 0;
}