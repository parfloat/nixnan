
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h> 
#include <hip/hip_fp16.h>
using namespace std;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {   
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }   
}
__global__ void half2_plus1(__half2 *in_array)
{
    const int idx = threadIdx.x + blockDim.x*blockIdx.x;
    // in_array[idx] = __hadd2(in_array[idx],  __float2half2_rn(1.0));
    in_array[idx] = __h2div(in_array[idx],  __float2half2_rn(1.0));
}
int main(void)
{
    const int n = 64;
    __half2 *h_in, *d_in;
    h_in = (__half2*) malloc(n*sizeof(__half2));
    gpuErrchk( hipMalloc(&d_in, n*sizeof(__half2)) );
    for (int i=0; i<n; i++)
        h_in[i] = __float2half2_rn(1.5);
    gpuErrchk( hipMemcpy(d_in, h_in, n*sizeof(__half2), hipMemcpyHostToDevice) );
    dim3 block_dims(2,1,1);
    dim3 thread_dims(32,1,1);
    half2_plus1<<<block_dims, thread_dims>>>(d_in);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpy(h_in, d_in, n*sizeof(__half2), hipMemcpyDeviceToHost) );
    for (int i=0; i<n; i++)
    {   
        if((__high2float(h_in[i]) != 2.5) || (__low2float(h_in[i]) !=2.5))
        {
            cout<< "Mismatch at " << i << " Expected = 2.5 " << "Actual = " << __half2float(h_in[i].x) << " " << __half2float(h_in[i].y) << endl;
            exit(1);
        }
        cout<< "Actual = " << __half2float(h_in[i].x) << " " << __half2float(h_in[i].y) << endl;
    }   
    cout << "TEST PASSES" << endl;
    hipFree(d_in);
    free(h_in);
    return 0;
}
