

#include <hip/hip_runtime.h>
#include <stdio.h>

// __device__ void mul(double a, double b, double *res)
// {
//   *res = a * b;
//   // NaN
//   *res = (*res)-(*res) / (*res)-(*res);
// }

__global__ void dot_prod(double *x, double *y, int size) {
  double d;
  for (int i = 0; i < size; ++i) {
    double tmp;
    // mul(x[i], y[i], &tmp);
    tmp = y[i] * double(0.0);
    //   printf("first tmp is %lf\n",tmp);
    // NaN
    // tmp = (tmp)-(tmp) / (tmp)-(tmp);
    // tmp = tmp / (tmp - tmp);
    tmp = x[i] / tmp;
    //    printf("second tmp is %lf\n",tmp);
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
