

#include <hip/hip_runtime.h>
#include <stdio.h>

// __device__ void mul(double a, double b, double *res)
// {
//   *res = a * b;
//   // Overflow

// #ifdef FPC_POSITIVE_OVERFLOW
//   *res = (*res) * 1e308;
// #else
//   *res = (*res) * 1e308;
// #endif
// }

__global__ void dot_prod(double *x, double *y, int size) {
  double d;
  for (int i = 0; i < size; ++i) {
    double tmp;
    // mul(x[i], y[i], &tmp);
    tmp = x[i] * y[i];
    // Overflow

    //  #ifdef FPC_POSITIVE_OVERFLOW
    //   tmp = (tmp) * 1e308;
    // #else
    tmp = tmp * 1e308;
    // #endif
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
