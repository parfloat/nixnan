#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>

// @min_sm 20

// rsqrt.approx{.ftz}.f64  d, a, b;

// .rnd = { .rn, .rz, .rm, .rp };

__global__ void test_rsqrt_approx(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("sqrt.rn.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}
__global__ void test_rsqrt_approx_ftz(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("sqrt.rn.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}


typedef void (*kernel_t)(const double*, double*);
int main() {
    int warpSize = 32;
    double *A, *B;
    kernel_t kernels[] = {test_rsqrt_approx, test_rsqrt_approx_ftz};
    hipMallocManaged(&A, warpSize * sizeof(double));
    hipMallocManaged(&B, warpSize * sizeof(double));
    double error_pairs[] = { 0.0, INFINITY, -1.0, -0.0, -INFINITY, -DBL_MIN/2 };
    for (kernel_t k : kernels) {
        for (auto a : error_pairs) {
            fill_array_double(A, warpSize, a);
            k<<<1, 32>>>(A, B);
            hipDeviceSynchronize();
        }
    }
    hipFree(A);
    hipFree(B);
    return 0;
}
