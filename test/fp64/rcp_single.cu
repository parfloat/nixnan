#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>

// @min_sm 20

// rcp{.rnd}.f64  d, a, b;

// .rnd = { .rn, .rz, .rm, .rp };

__global__ void test_rcp_rn(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("rcp.rn.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}
__global__ void test_rcp_rz(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("rcp.rz.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}
__global__ void test_rcp_rm(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("rcp.rm.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}
__global__ void test_rcp_rp(const double* a, double* b) {
    int idx = threadIdx.x;
    asm ("rcp.rp.f64 %0, %1;" : "=d"(b[idx]) : "d"(a[idx]));
}

typedef void (*kernel_t)(const double*, double*);
int main() {
    int warpSize = 32;
    double *A, *B;
    kernel_t kernels[] = {test_rcp_rn, test_rcp_rz, test_rcp_rm, test_rcp_rp};
    hipMallocManaged(&A, warpSize * sizeof(double));
    hipMallocManaged(&B, warpSize * sizeof(double));
    double error_pairs[] = { 0.0, INFINITY, -INFINITY, DBL_MIN, DBL_MAX };
    for (kernel_t k : kernels) {
        for (auto a : error_pairs) {
            fill_array_double(A, warpSize, a);
            k<<<1, 32>>>(A, B);
            hipDeviceSynchronize();
        }
    }
    hipFree(A);
    hipFree(B);
    return 0;
}
