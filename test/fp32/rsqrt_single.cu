#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>

// @min_sm 20

// rsqrt.approx{.ftz}.f32  d, a; // fast, approximate square root

// .rnd = { .rn, .rz, .rm, .rp };

__global__ void test_rsqrt_approx_(const float* a, float* b) {
    int idx = threadIdx.x;
    asm ("rsqrt.approx.f32 %0, %1;" : "=f"(b[idx]) : "f"(a[idx]));
}
__global__ void test_rsqrt_approx_ftz(const float* a, float* b) {
    int idx = threadIdx.x;
    asm ("rsqrt.approx.ftz.f32 %0, %1;" : "=f"(b[idx]) : "f"(a[idx]));
}

typedef void (*kernel_t)(const float*, float*);
int main() {
    int warpSize = 32;
    float *A, *B;
    kernel_t kernels[] = {test_rsqrt_approx_, test_rsqrt_approx_ftz};
    hipMallocManaged(&A, warpSize * sizeof(float));
    hipMallocManaged(&B, warpSize * sizeof(float));
    float vals[] = {-1.0, FLT_MIN, 0.0, -0.0, INFINITY};
    for (kernel_t k : kernels) {
        for (float val : vals) {
            fill_array_float(A, warpSize, val);
            k<<<1, 32>>>(A, B);
            hipDeviceSynchronize();
        }
    }
    hipFree(A);
    hipFree(B);
    return 0;
}
