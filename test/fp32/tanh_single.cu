#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>

// @min_sm 20

__global__ void test_tanh_approx_(const float* a, float* b) {
    int idx = threadIdx.x;
    asm ("tanh.approx.f32 %0, %1;" : "=f"(b[idx]) : "f"(a[idx]));
}

typedef void (*kernel_t)(const float*, float*);
int main() {
    int warpSize = 32;
    float *A, *B;
    kernel_t kernels[] = {test_tanh_approx_};
    hipMallocManaged(&A, warpSize * sizeof(float));
    hipMallocManaged(&B, warpSize * sizeof(float));
    float vals[] = {-1.0, FLT_MIN, 0.0, -0.0, INFINITY, NAN};
    for (kernel_t k : kernels) {
        for (float val : vals) {
            fill_array_float(A, warpSize, val);
            k<<<1, 32>>>(A, B);
            hipDeviceSynchronize();
            for (int i = 0; i < warpSize; i++) {
                printf("tanh.approx(%e) = %e\n", A[i], B[i]);
            }
        }
    }
    hipFree(A);
    hipFree(B);
    return 0;
}
