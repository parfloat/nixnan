#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>
#include <hip/hip_fp16.h>

// @min_sm 20

// sub{.rnd}{.ftz}{.sat}.f16   d, a, b;
// sub{.rnd}{.ftz}{.sat}.f16x2 d, a, b;

// .rnd = { .rn };

__global__ void test_fma___(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma___sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma__ftz_(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.ftz.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma__ftz_sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.ftz.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn__(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.rn.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn__sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.rn.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn_ftz_(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.rn.ftz.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn_ftz_sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("sub.rn.ftz.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}

typedef void (*kernel_t)(const half*, const half*, half*);
int main() {
    int warpSize = 32;
    half *A, *B, *C;
    kernel_t kernels[] = {test_fma___, test_fma___sat, test_fma__ftz_, test_fma__ftz_sat,
                          test_fma_rn__, test_fma_rn__sat, test_fma_rn_ftz_,
                          test_fma_rn_ftz_sat};
    hipMallocManaged(&A, warpSize * sizeof(half));
    hipMallocManaged(&B, warpSize * sizeof(half));
    hipMallocManaged(&C, warpSize * sizeof(half));
    std::pair<half, half> error_pairs[] = {
        {INFINITY, INFINITY},
        {HLF_MAX, -HLF_MAX},
        {HLF_MIN, HLF_MIN/2.0}};
    for (kernel_t k : kernels) {
        for (auto [a, b] : error_pairs) {
            fill_array_half(A, warpSize, a);
            fill_array_half(B, warpSize, b);
            k<<<1, 32>>>(A, B, C);
            hipDeviceSynchronize();
        }
    }
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}
