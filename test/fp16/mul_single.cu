#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "../helpers.cuh"
#include <cfloat>
#include <hip/hip_fp16.h>

// @min_sm 20

// sub{.rnd}{.ftz}{.sat}.f16   d, a, b;
// sub{.rnd}{.ftz}{.sat}.f16x2 d, a, b;

// .rnd = { .rn };

__global__ void test_fma___(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma___sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma__ftz_(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.ftz.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma__ftz_sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.ftz.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn__(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.rn.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn__sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.rn.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn_ftz_(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.rn.ftz.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}
__global__ void test_fma_rn_ftz_sat(const half* a, const half* b, half* c) {
    int idx = threadIdx.x;
    short out;
    asm ("mul.rn.ftz.sat.f16 %0, %1, %2;" : "=h"(out) : "h"(h2s(a[idx])), "h"(h2s(b[idx])));
    c[idx] = s2h(out);
}

typedef void (*kernel_t)(const half*, const half*, half*);
int main() {
    int warpSize = 32;
    half *A, *B, *C;
    kernel_t kernels[] = {test_fma___, test_fma___sat, test_fma__ftz_, test_fma__ftz_sat,
                          test_fma_rn__, test_fma_rn__sat, test_fma_rn_ftz_,
                          test_fma_rn_ftz_sat};
    hipMallocManaged(&A, warpSize * sizeof(half));
    hipMallocManaged(&B, warpSize * sizeof(half));
    hipMallocManaged(&C, warpSize * sizeof(half));
    std::pair<half, half> error_pairs[] = {
        {INFINITY, 0},
        {HLF_MAX, HLF_MAX},
        {HLF_MIN, 0.5}};
    for (kernel_t k : kernels) {
        for (auto [a, b] : error_pairs) {
            fill_array_half(A, warpSize, a);
            fill_array_half(B, warpSize, b);
            k<<<1, 32>>>(A, B, C);
            hipDeviceSynchronize();
        }
    }
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}
